#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/detail/get_value.cuh>
#include <cudf/detail/null_mask.hpp>
#include <cudf/detail/nvtx/ranges.hpp>
#include <cudf/detail/utilities/cuda.cuh>
#include <cudf/strings/convert/convert_urls.hpp>
#include <cudf/strings/detail/utilities.cuh>
#include <cudf/strings/detail/utilities.hpp>
#include <cudf/strings/string_view.cuh>
#include <cudf/strings/strings_column_view.hpp>
#include <cudf/utilities/span.hpp>

#include <rmm/cuda_stream_view.hpp>
#include <rmm/device_uvector.hpp>

#include <hipcub/hipcub.hpp>

#include <thrust/binary_search.h>
#include <thrust/count.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/scan.h>

#include <algorithm>
#include <iostream>

using cudf::device_span;

namespace cudf {
namespace strings {
namespace detail {
namespace {
//
// This is the functor for the url_encode() method below.
// Specific requirements are documented in custrings issue #321.
// In summary it converts mostly non-ascii characters and control characters into UTF-8 hex
// characters prefixed with '%'. For example, the space character must be converted to characters
// '%20' where the '20' indicates the hex value for space in UTF-8. Likewise, multi-byte characters
// are converted to multiple hex characters. For example, the é character is converted to characters
// '%C3%A9' where 'C3A9' is the UTF-8 bytes xc3a9 for this character.
//
struct url_encoder_fn {
  column_device_view const d_strings;
  int32_t const* d_offsets{};
  char* d_chars{};

  // utility to create 2-byte hex characters from single binary byte
  __device__ void byte_to_hex(uint8_t byte, char* hex)
  {
    hex[0] = '0';
    if (byte >= 16) {
      uint8_t hibyte = byte / 16;
      hex[0]         = hibyte < 10 ? '0' + hibyte : 'A' + (hibyte - 10);
      byte           = byte - (hibyte * 16);
    }
    hex[1] = byte < 10 ? '0' + byte : 'A' + (byte - 10);
  }

  __device__ bool should_not_url_encode(char ch)
  {
    return (
      (ch >= '0' && ch <= '9') ||  // these are the characters
      (ch >= 'A' && ch <= 'Z') ||  // that are not to be url encoded
      (ch >= 'a' &&
       ch <= 'z') ||  // reference: docs.python.org/3/library/urllib.parse.html#urllib.parse.quote
      (ch == '.') ||
      (ch == '_') || (ch == '~') || (ch == '-'));
  }

  // main part of the functor the performs the url-encoding
  __device__ size_type operator()(size_type idx)
  {
    if (d_strings.is_null(idx)) return 0;
    string_view d_str = d_strings.element<string_view>(idx);
    //
    char* out_ptr    = d_chars ? d_chars + d_offsets[idx] : nullptr;
    size_type nbytes = 0;
    char hex[2];  // two-byte hex max
    for (auto itr = d_str.begin(); itr != d_str.end(); ++itr) {
      auto ch = *itr;
      if (ch < 128) {
        if (should_not_url_encode(static_cast<char>(ch))) {
          nbytes++;
          if (out_ptr) out_ptr = copy_and_increment(out_ptr, d_str.data() + itr.byte_offset(), 1);
        } else  // url-encode everything else
        {
          nbytes += 3;
          if (out_ptr) {
            out_ptr = copy_and_increment(out_ptr, "%", 1);  // add the '%' prefix
            byte_to_hex(static_cast<uint8_t>(ch), hex);     // convert to 2 hex chars
            out_ptr = copy_and_increment(out_ptr, hex, 2);  // add them to the output
          }
        }
      } else  // these are to be utf-8 url-encoded
      {
        uint8_t char_bytes[4];  // holds utf-8 bytes for one character
        size_type char_width = from_char_utf8(ch, reinterpret_cast<char*>(char_bytes));
        nbytes += char_width * 3;  // '%' plus 2 hex chars per byte (example: é is %C3%A9)
        // process each byte in this current character
        for (size_type chidx = 0; out_ptr && (chidx < char_width); ++chidx) {
          out_ptr = copy_and_increment(out_ptr, "%", 1);  // add '%' prefix
          byte_to_hex(char_bytes[chidx], hex);            // convert to 2 hex chars
          out_ptr = copy_and_increment(out_ptr, hex, 2);  // add them to the output
        }
      }
    }
    return nbytes;
  }
};

}  // namespace

//
std::unique_ptr<column> url_encode(
  strings_column_view const& strings,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(data_type{type_id::STRING});

  auto strings_column = column_device_view::create(strings.parent(), stream);
  auto d_strings      = *strings_column;

  // copy null mask
  rmm::device_buffer null_mask = cudf::detail::copy_bitmask(strings.parent(), stream, mr);
  // build offsets column
  auto offsets_transformer_itr = thrust::make_transform_iterator(
    thrust::make_counting_iterator<size_type>(0), url_encoder_fn{d_strings});
  auto offsets_column = make_offsets_child_column(
    offsets_transformer_itr, offsets_transformer_itr + strings_count, stream, mr);
  auto d_offsets = offsets_column->view().data<int32_t>();
  auto const bytes =
    cudf::detail::get_value<int32_t>(offsets_column->view(), strings_count, stream);
  // build chars column
  auto chars_column = create_chars_child_column(bytes, stream, mr);
  auto d_chars      = chars_column->mutable_view().data<char>();
  thrust::for_each_n(rmm::exec_policy(stream),
                     thrust::make_counting_iterator<size_type>(0),
                     strings_count,
                     url_encoder_fn{d_strings, d_offsets, d_chars});

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             strings.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API
std::unique_ptr<column> url_encode(strings_column_view const& strings,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::url_encode(strings, rmm::cuda_stream_default, mr);
}

namespace detail {
namespace {

// utility to convert a hex char into a single byte
constexpr uint8_t hex_char_to_byte(char ch)
{
  if (ch >= '0' && ch <= '9') return (ch - '0');
  if (ch >= 'A' && ch <= 'F') return (ch - 'A' + 10);  // in hex A=10,B=11,...,F=15
  if (ch >= 'a' && ch <= 'f') return (ch - 'a' + 10);  // same for lower case
  return 0;
}

constexpr bool is_hex_digit(char ch)
{
  return (ch >= '0' && ch <= '9') || (ch >= 'A' && ch <= 'F') || (ch >= 'a' && ch <= 'f');
}

__forceinline__ __device__ bool is_escape_char(char const* const ptr)
{
  return (ptr[0] == '%' && is_hex_digit(ptr[1]) && is_hex_digit(ptr[2]));
}

template <int warps_per_threadblock, int block_size>
__global__ void url_decode_char_counter(char const* const in_chars,
                                        int32_t const* const in_offsets,
                                        int32_t* const out_counts,
                                        int32_t const num_rows)
{
  __shared__ char temporary_buffer[warps_per_threadblock][block_size + 2];
  __shared__ typename hipcub::WarpReduce<int8_t>::TempStorage cub_storage[warps_per_threadblock];

  int global_thread_id  = blockIdx.x * blockDim.x + threadIdx.x;
  int global_warp_id    = global_thread_id / cudf::detail::warp_size;
  int local_warp_id     = threadIdx.x / cudf::detail::warp_size;
  int warp_lane         = threadIdx.x % cudf::detail::warp_size;
  int nwarps            = gridDim.x * blockDim.x / cudf::detail::warp_size;
  char* in_chars_shared = temporary_buffer[local_warp_id];

  for (size_type row_idx = global_warp_id; row_idx < num_rows; row_idx += nwarps) {
    auto in_chars_string      = in_chars + in_offsets[row_idx];
    auto string_length        = in_offsets[row_idx + 1] - in_offsets[row_idx];
    int nblocks               = (string_length + block_size - 1) / block_size;
    int32_t escape_char_count = 0;

    for (int iblock = 0; iblock < nblocks; iblock++) {
      int string_length_iblock = std::min(block_size, string_length - block_size * iblock);

      // Each warp collectively loads input characters of the current block to shared memory
      for (int ichar = warp_lane; ichar < string_length_iblock + 2;
           ichar += cudf::detail::warp_size) {
        char ch    = 0;
        int in_idx = iblock * block_size + ichar;
        if (in_idx < string_length) ch = in_chars_string[in_idx];
        in_chars_shared[ichar] = ch;
      }

      __syncwarp();

      for (int ichar_start = 0; ichar_start < string_length_iblock;
           ichar_start += cudf::detail::warp_size) {
        int ichar = ichar_start + warp_lane;
        int8_t is_ichar_escape_char =
          (ichar < string_length_iblock && is_escape_char(in_chars_shared + ichar)) ? 1 : 0;

        int8_t total_escape_char =
          hipcub::WarpReduce<int8_t>(cub_storage[local_warp_id]).Sum(is_ichar_escape_char);

        if (warp_lane == 0) { escape_char_count += total_escape_char; }

        __syncwarp();
      }
    }
    if (warp_lane == 0) { out_counts[row_idx] = string_length - escape_char_count * 2; }
  }
}

template <int warps_per_threadblock, int block_size>
__global__ void url_decode_char_replacer(char const* const in_chars,
                                         int32_t const* const in_offsets,
                                         char* const out_chars,
                                         int32_t const* const out_offsets,
                                         int32_t const num_rows)
{
  __shared__ char temporary_buffer[warps_per_threadblock][block_size + 4];
  __shared__ typename hipcub::WarpScan<int8_t>::TempStorage cub_storage[warps_per_threadblock];
  __shared__ int out_idx[warps_per_threadblock];

  int global_thread_id  = blockIdx.x * blockDim.x + threadIdx.x;
  int global_warp_id    = global_thread_id / cudf::detail::warp_size;
  int local_warp_id     = threadIdx.x / cudf::detail::warp_size;
  int warp_lane         = threadIdx.x % cudf::detail::warp_size;
  int nwarps            = gridDim.x * blockDim.x / cudf::detail::warp_size;
  char* in_chars_shared = temporary_buffer[local_warp_id];

  for (size_type row_idx = global_warp_id; row_idx < num_rows; row_idx += nwarps) {
    auto in_chars_string  = in_chars + in_offsets[row_idx];
    auto out_chars_string = out_chars + out_offsets[row_idx];
    auto string_length    = in_offsets[row_idx + 1] - in_offsets[row_idx];
    int nblocks           = (string_length + block_size - 1) / block_size;

    for (int iblock = 0; iblock < nblocks; iblock++) {
      int string_length_iblock = std::min(block_size, string_length - block_size * iblock);

      // Each warp collectively loads input characters of the current block to shared memory
      for (int ichar = warp_lane; ichar < string_length_iblock + 4;
           ichar += cudf::detail::warp_size) {
        char ch    = 0;
        int in_idx = iblock * block_size + ichar - 2;
        if (in_idx >= 0 && in_idx < string_length) ch = in_chars_string[in_idx];
        in_chars_shared[ichar] = ch;
      }

      if (warp_lane == cudf::detail::warp_size - 1) { out_idx[local_warp_id] = 0; }

      __syncwarp();

      for (int ichar_start = 0; ichar_start < string_length_iblock;
           ichar_start += cudf::detail::warp_size) {
        int ichar = ichar_start + warp_lane;
        int8_t out_size =
          (ichar >= string_length_iblock || is_escape_char(in_chars_shared + ichar) ||
           is_escape_char(in_chars_shared + ichar + 1))
            ? 0
            : 1;
        int8_t out_offset;

        hipcub::WarpScan<int8_t>(cub_storage[local_warp_id]).ExclusiveSum(out_size, out_offset);

        if (out_size == 1) {
          char ch;
          if (is_escape_char(in_chars_shared + ichar + 2)) {
            ch = (hex_char_to_byte(in_chars_shared[ichar + 3]) << 4) |
                 hex_char_to_byte(in_chars_shared[ichar + 4]);
          } else {
            ch = in_chars_shared[ichar + 2];
          }
          out_chars_string[out_idx[local_warp_id] + out_offset] = ch;
        }

        if (warp_lane == cudf::detail::warp_size - 1) {
          out_idx[local_warp_id] += (out_offset + out_size);
        };

        __syncwarp();
      }
    }
  }
}

}  // namespace

//
std::unique_ptr<column> url_decode(
  strings_column_view const& strings,
  rmm::cuda_stream_view stream,
  rmm::mr::device_memory_resource* mr = rmm::mr::get_current_device_resource())
{
  size_type strings_count = strings.size();
  if (strings_count == 0) return make_empty_column(data_type{type_id::STRING});

  auto offset_count = strings_count + 1;
  auto d_offsets    = strings.offsets().data<int32_t>() + strings.offset();
  auto d_in_chars   = strings.chars().data<char>();

  /*
  if (esc_count == 0) {
    // nothing to replace, so just copy the input column
    return std::make_unique<cudf::column>(strings.parent(), stream, mr);
  }
  */

  // build offsets column
  auto offsets_column = make_numeric_column(
    data_type{type_id::INT32}, offset_count, mask_state::UNALLOCATED, stream, mr);

  // count number of bytes in each row after decoding
  auto offsets_view         = offsets_column->view();
  auto offsets_mutable_view = offsets_column->mutable_view();
  url_decode_char_counter<4, 286><<<65536, 128, 0, stream.value()>>>(
    d_in_chars, d_offsets, offsets_mutable_view.begin<int32_t>() + 1, strings_count);

  thrust::inclusive_scan(rmm::exec_policy(stream),
                         offsets_view.begin<int32_t>() + 1,
                         offsets_view.begin<int32_t>() + offset_count,
                         offsets_mutable_view.begin<int32_t>() + 1);

  CUDA_TRY(
    hipMemsetAsync(offsets_mutable_view.begin<int32_t>(), 0, sizeof(int32_t), stream.value()));

  int32_t out_chars_bytes;
  CUDA_TRY(hipMemcpyAsync(&out_chars_bytes,
                           offsets_view.begin<int32_t>() + offset_count - 1,
                           sizeof(int32_t),
                           hipMemcpyDeviceToHost,
                           stream.value()));
  stream.synchronize();

  // create the chars column
  auto chars_column = create_chars_child_column(out_chars_bytes, stream, mr);
  auto d_out_chars  = chars_column->mutable_view().data<char>();

  url_decode_char_replacer<4, 284><<<65536, 128, 0, stream.value()>>>(
    d_in_chars, d_offsets, d_out_chars, offsets_column->view().begin<int32_t>(), strings_count);

  // copy null mask
  rmm::device_buffer null_mask = cudf::detail::copy_bitmask(strings.parent(), stream, mr);

  return make_strings_column(strings_count,
                             std::move(offsets_column),
                             std::move(chars_column),
                             strings.null_count(),
                             std::move(null_mask),
                             stream,
                             mr);
}

}  // namespace detail

// external API

std::unique_ptr<column> url_decode(strings_column_view const& strings,
                                   rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE();
  return detail::url_decode(strings, rmm::cuda_stream_default, mr);
}

}  // namespace strings
}  // namespace cudf
